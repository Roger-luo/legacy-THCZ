#include "THCZBlas.h"
#include "THCZGeneral.h"

hipComplex THCudaBlas_Cdotc(THCState *state, int64_t n, hipComplex *x, int64_t incx, hipComplex *y, int64_t incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    hipComplex result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasCdotc(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Cdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return hipComplex();
}


hipDoubleComplex THCudaBlas_Zdotc(THCState *state, int64_t n, hipDoubleComplex *x, int64_t incx, hipDoubleComplex *y, int64_t incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    hipDoubleComplex result;
    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasZdotc(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Zdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return hipDoubleComplex();
}


/* Level 2 */
void THCudaBlas_Cgemv(THCState *state, char trans, int64_t m, int64_t n, hipComplex alpha, hipComplex *a, int64_t lda, hipComplex *x, int64_t incx, hipComplex beta, hipComplex *y, int64_t incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasCgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Cgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}


void THCudaBlas_Zgemv(THCState *state, char trans, int64_t m, int64_t n, hipDoubleComplex alpha, hipDoubleComplex *a, int64_t lda, hipDoubleComplex *x, int64_t incx, hipDoubleComplex beta, hipDoubleComplex *y, int64_t incy)
{
  if(n == 1)
    lda = m;

  hipblasOperation_t op;
  if (trans == 't') op = HIPBLAS_OP_T;
  else if (trans == 'n') op = HIPBLAS_OP_N;
  else if (trans == 'c') op = HIPBLAS_OP_C;

  if( (m <= INT_MAX) && (n <= INT_MAX) &&
      (lda > 0) && (lda <= INT_MAX) &&
      (incx > 0) && (incx <= INT_MAX) &&
      (incy > 0) && (incy <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_lda = (int)lda;
    int i_incx = (int)incx;
    int i_incy = (int)incy;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasZgemv(handle, op, i_m, i_n, &alpha, a, i_lda, x, i_incx, &beta, y, i_incy));
    return;
  }
  THError("Cublas_Zgemv only supports m, n, lda, incx, incy"
          "in the range 0 < [val] <= %d", INT_MAX);
}

void THCudaBlas_Cgerc(THCState *state, int64_t m, int64_t n, hipComplex alpha, hipComplex *x, int64_t incx, hipComplex *y, int64_t incy, hipComplex *a, int64_t lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasCgerc(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Cgerc only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}


void THCudaBlas_Zgerc(THCState *state, int64_t m, int64_t n, hipDoubleComplex alpha, hipDoubleComplex *x, int64_t incx, hipDoubleComplex *y, int64_t incy, hipDoubleComplex *a, int64_t lda)
{
  if(n == 1)
    lda = m;

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
      hipblasSetStream(handle, THCState_getCurrentStream(state));
      THCublasCheck(hipblasZgerc(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Zgerc only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}


/* Level 3 */
void THCudaBlas_Cgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, hipComplex alpha, hipComplex *a, int64_t lda, hipComplex *b, int64_t ldb, hipComplex beta, hipComplex *c, int64_t ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasCgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Cgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}


void THCudaBlas_Zgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, hipDoubleComplex alpha, hipDoubleComplex *a, int64_t lda, hipDoubleComplex *b, int64_t ldb, hipDoubleComplex beta, hipDoubleComplex *c, int64_t ldc)
{
  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (k <= INT_MAX) && (lda <= INT_MAX)  && (ldb <= INT_MAX) && (ldc <= INT_MAX) )
  {
    int i_m = (int)m;
    int i_n = (int)n;
    int i_k = (int)k;
    int i_lda = (int)lda;
    int i_ldb = (int)ldb;
    int i_ldc = (int)ldc;

    hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
    hipblasSetStream(handle, THCState_getCurrentStream(state));
    THCublasCheck(hipblasZgemm(handle, opa, opb, i_m, i_n, i_k, &alpha, a, i_lda, b, i_ldb, &beta, c, i_ldc));
    return;
  }
  THError("Cublas_Zgemm only supports m, n, k, lda, ldb, ldc"
          "with the bound [val] <= %d", INT_MAX);
}


void THCudaBlas_CgemmBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                           hipComplex alpha, const hipComplex *a[], int64_t lda, const hipComplex *b[], int64_t ldb,
                           hipComplex beta, hipComplex *c[], int64_t ldc, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_CgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasCgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
}


#if CUDA_VERSION >= 8000
void THCudaBlas_CgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                           hipComplex alpha, const hipComplex *a, int64_t lda, int64_t strideA, const hipComplex *b, int64_t ldb, int64_t strideB,
                           hipComplex beta, hipComplex *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_CgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasCgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif


void THCudaBlas_ZgemmBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             hipDoubleComplex alpha, const hipDoubleComplex *a[], int64_t lda, const hipDoubleComplex *b[], int64_t ldb,
                             hipDoubleComplex beta, hipDoubleComplex *c[], int64_t ldc, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_ZgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasZgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
}

#if CUDA_VERSION >= 8000
void THCudaBlas_ZgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                                 hipDoubleComplex alpha, const hipDoubleComplex *a, int64_t lda, int64_t strideA, const hipDoubleComplex *b, int64_t ldb, int64_t strideB, 
                                 hipDoubleComplex beta, hipDoubleComplex *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_ZgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLd(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasZgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC, 
                                   (int)batchCount));
}
#endif

/* Inverse */
void THCudaBlas_Cgetrf(THCState *state, int n, hipComplex **a, int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Cgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasCgetrfBatched(handle, n, a, lda, pivot, info, batchSize));
}

void THCudaBlas_Zgetrf(THCState *state, int n, hipDoubleComplex **a, int lda, int *pivot, int *info, int batchSize) {
  if( (n >= INT_MAX) || (lda >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Zgetrf only supports n, lda, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasZgetrfBatched(handle, n, a, lda, pivot, info, batchSize));
}

void THCudaBlas_Cgetrs(THCState *state, char transa, int n, int nrhs, const hipComplex **a, int lda, int *pivot, hipComplex **b, int ldb, int *info, int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Cgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasCgetrsBatched(handle, opa, n, nrhs, a, lda, pivot, b, ldb, info, batchSize));
}


void THCudaBlas_Zgetrs(THCState *state, char transa, int n, int nrhs, const hipDoubleComplex **a, int lda, int *pivot, hipDoubleComplex **b, int ldb, int *info, int batchSize)
{
  if( (n >= INT_MAX) || (nrhs >= INT_MAX) || (lda >= INT_MAX) || (ldb >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Zgetrs only supports n, nrhs, lda, ldb, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }

  // no need to adjust leading dimensions, since matrices are square
  hipblasOperation_t opa = convertTransToCublasOperation(transa);

  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasZgetrsBatched(handle, opa, n, nrhs, a, lda, pivot, b, ldb, info, batchSize));
}

void THCudaBlas_Cgetri(THCState *state, int n, const hipComplex **a, int lda, int *pivot, hipComplex **c, int ldc, int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Cgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasCgetriBatched(handle, n, a, lda, pivot, c, ldc, info, batchSize));
}

void THCudaBlas_Zgetri(THCState *state, int n, const hipDoubleComplex **a, int lda, int *pivot, hipDoubleComplex **c, int ldc, int *info, int batchSize) {

  if( (n >= INT_MAX) || (lda >= INT_MAX)|| (ldc >= INT_MAX) || (batchSize >= INT_MAX) )
  {
    THError("Cublas_Zgetri only supports n, lda, ldc, batchSize"
            "with the bound [val] <= %d", INT_MAX);
  }
  hipblasHandle_t handle = THCState_getCurrentBlasHandle(state);
  hipblasSetStream(handle, THCState_getCurrentStream(state));
  THCublasCheck(hipblasZgetriBatched(handle, n, a, lda, pivot, c, ldc, info, batchSize));
}

